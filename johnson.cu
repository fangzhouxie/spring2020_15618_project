#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cycletimer.hpp"
#include "instrument.hpp"

#define MaxLineLength 1024
#define IntMax __INT32_MAX__
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

char display;

struct GlobalConstants {
    int nnode;
    int nedge;

    int *node;
    int *edge;
    int *weight;
};

typedef struct {
    int nnode;
    int nedge;

    int *node;
    int *edge;
    int *weight;
    int *new_weight;

    int *distance;
    int *predecessor;
} Graph;

__constant__ GlobalConstants constGraphParams;

Graph *LoadGraph(FILE *graph_file) {
    Graph *graph = (Graph *)malloc(sizeof(Graph));
    char linebuf[MaxLineLength];
    int src_id, dst_id, weight;
    int prev_src_id = 0;
    int lineno = 0;

    //  Load number of nodes and edges
    fgets(linebuf, MaxLineLength, graph_file);
    if (sscanf(linebuf, "%d", &graph->nnode) < 1) {
        printf("ERROR. Malformed graph file header (line 1)\n");
        return NULL;
    }

    fgets(linebuf, MaxLineLength, graph_file);
    if (sscanf(linebuf, "%d", &graph->nedge) < 1) {
        printf("ERROR. Malformed graph file header (line 1)\n");
        return NULL;
    }

    //  Initialize graph
    graph->node = (int *)calloc(graph->nnode+1, sizeof(int));
    graph->node[graph->nnode] = graph->nedge;
    graph->edge = (int *)malloc(graph->nedge * sizeof(int));
    graph->weight = (int *)malloc(graph->nedge * sizeof(int));
    graph->new_weight = (int *)malloc(graph->nedge * sizeof(int));
    graph->distance = (int *)malloc(graph->nnode * graph->nnode * sizeof(int));
    graph->predecessor = (int *)malloc(graph->nnode * graph->nnode * sizeof(int));

    //  Load edges
    while (fgets(linebuf, MaxLineLength, graph_file) != NULL) {
        if (sscanf(linebuf, "%d %d %d", &src_id, &dst_id, &weight) < 3) {
            printf("ERROR. Malformed graph file header (line 1)\n");
            return NULL;
        }

        if (prev_src_id != src_id) {
            for (int i = prev_src_id+1; i <= src_id; i++)
                graph->node[i] = lineno;
            prev_src_id = src_id;
        }

        graph->edge[lineno] = dst_id;
        graph->weight[lineno] = weight;

        lineno++;
    }

    // Pad all later nodes with 0 out degree
    for (int nid = src_id+1; nid < graph->nnode; nid++) graph->node[nid] = graph->nedge;

    return graph;
}

void freeGraph(Graph* graph) {
    free(graph->node);
    free(graph->edge);
    free(graph->weight);
    free(graph->new_weight);
    free(graph->distance);
    free(graph->predecessor);
    free(graph);
}

///////////////////////////////////////////////////////////////////////////////
// Start of kernels
///////////////////////////////////////////////////////////////////////////////

__device__ __inline__ void BellmanFord(Graph *graph, int nid) {
    extern __shared__ int distance[];
    if (nid < graph->nnode)
        distance[nid] = 0;

    __syncthreads();

    for (int u = 0; u < graph->nnode; u++)
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            int weight = graph->weight[eid];
            if (distance[v] > distance[u] + weight)
                distance[v] = distance[u] + weight;
        }

    __syncthreads();

    if (nid < graph->nnode) {
        int u = nid;
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            graph->new_weight[eid] = graph->weight[eid] + distance[u] - distance[v];
            if (graph->new_weight[eid] < 0) {
                printf("Graph contains negative weight cycle\n");
                return;
            }
        }
    }
}

// Recursively calculate original weights
__device__ __inline__ void CalculateOriginalDistance(int src_nid, int nid, int *distance, int *predecessor) {
    int current_nid = nid;
    int prev_nid = predecessor[current_nid];

    int *node = constGraphParams.node;
    int *edge = constGraphParams.edge;
    int *weight = constGraphParams.weight;

    if (distance[nid] != -1)   // Distance is already alculated
        return;
    else if (nid == src_nid)    // This is the source node
        distance[nid] = 0;
    else if (predecessor[nid] == -1)    // No valid path to this node exists
        distance[nid] = IntMax;
    else {
        if (distance[prev_nid] == -1)
            CalculateOriginalDistance(src_nid, prev_nid, distance, predecessor);
        // Distance increment by original edge weight
        for (int eid = node[prev_nid]; eid < node[prev_nid+1]; eid++)
            if (edge[eid] == current_nid)
                distance[nid] = weight[eid] + distance[prev_nid];
    }
}

// Functionality is explaned by function name
__device__ __inline__ int FindIndexOfUnvisitedNodeWithMinDistance(int nnode, int *distance, char *visited) {
    int min_nid = -1;
    int min_distance = IntMax;

    for (int nid = 0; nid < nnode; nid++)
        if (!visited[nid] && distance[nid] <= min_distance) {
            min_nid = nid;
            min_distance = distance[nid];
        }

    return min_nid;
}

__global__ void dijkstra_kernel(int* new_weight, int* distance, int* tmp_distance, int* predecessor, char* visited) {
    int src_nid = blockIdx.x * blockDim.x + threadIdx.x;
    int nnode = constGraphParams.nnode;
    if (src_nid >= nnode) return;

    int *node = constGraphParams.node;
    int *edge = constGraphParams.edge;

    int *distance_local = &distance[src_nid * nnode];
    // malloc would fail for large graphs
    int *predecessor_local = &predecessor[src_nid * nnode];
    int *tmp_distance_local = &tmp_distance[src_nid * nnode];
    char *visited_local = &visited[src_nid * nnode];

    for (int nid = 0; nid < nnode; nid++) {
        distance_local[nid] = -1;
        predecessor_local[nid] = -1;
        tmp_distance_local[nid] = IntMax;
        visited_local[nid] = 0;
    }
    tmp_distance_local[src_nid] = 0;
    predecessor_local[src_nid] = src_nid;

    for (int iter = 0; iter < nnode; iter++) {
        int min_nid = FindIndexOfUnvisitedNodeWithMinDistance(nnode, tmp_distance_local, visited_local);
        // No reachable unvisted nodes left
        if (tmp_distance_local[min_nid] == IntMax) break;

        visited_local[min_nid] = 1;
        for (int eid = node[min_nid]; eid < node[min_nid+1]; eid++) {
            int neighbor_nid = edge[eid];
            if (tmp_distance_local[neighbor_nid] > new_weight[eid] + tmp_distance_local[min_nid]) {
                tmp_distance_local[neighbor_nid] = new_weight[eid] + tmp_distance_local[min_nid];
                predecessor_local[neighbor_nid] = min_nid;
            }
        }
    }

    for (int nid = 0; nid < nnode; nid++)
        CalculateOriginalDistance(src_nid, nid, distance_local, predecessor_local);

}

__global__ void bellman_ford_kernel(int* src_nodes, int* dst_nodes, int* distance) {
    int eid = blockIdx.x * blockDim.x + threadIdx.x;
    int nedge = constGraphParams.nedge;
    if (eid >= nedge) return;

    int u = src_nodes[eid];
    int v = dst_nodes[eid];

    int* weight = constGraphParams.weight;

    int new_distance = distance[u] + weight[eid];

    // do we need atomicCAS?
    //if (distance[v] > new_distance) distance[v] = new_distance;
    atomicMin(&distance[v], new_distance);
}

///////////////////////////////////////////////////////////////////////////////
/// End of kernels
///////////////////////////////////////////////////////////////////////////////

__host__ void bellman_ford_host(Graph *graph) {
    int distance[graph->nnode];

    // Initialize distances from new source node to all nodes
    for (int nid = 0; nid < graph->nnode; nid++)
        distance[nid] = 0;

    /**************************************************************/
    int* srcNodes = (int *)malloc(graph->nedge * sizeof(int));
    int* dstNodes = (int *)malloc(graph->nedge * sizeof(int));

    for (int u=0; u < graph->nnode; u++) {
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            srcNodes[eid] = u;
            dstNodes[eid] = v;
        }
    }

    // Iterate through the graph V - 1 times
    int threadsPerBlock = 32;
    int blocks = (graph->nedge + threadsPerBlock - 1) / threadsPerBlock;

    int* deviceSrcNodes;
    int* deviceDstNodes;
    int* deviceDistance;

    hipMalloc(&deviceSrcNodes, graph->nedge * sizeof(int));
    hipMalloc(&deviceDstNodes, graph->nedge * sizeof(int));
    hipMalloc(&deviceDistance, graph->nnode * sizeof(int));
    // cudaCheckErrors("bellman_ford hipMalloc");

    hipMemcpy(deviceSrcNodes, srcNodes, graph->nedge * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceDstNodes, dstNodes, graph->nedge * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceDistance, distance, graph->nnode * sizeof(int), hipMemcpyHostToDevice);
    // cudaCheckErrors("bellman_ford hipMemcpyHostToDevice");

    for (int iter = 0; iter < graph->nnode; iter++) {
        bellman_ford_kernel<<<blocks, threadsPerBlock>>>(deviceSrcNodes, deviceDstNodes, deviceDistance);
        hipDeviceSynchronize(); // sync before next iteration
    }

    // cudaCheckErrors("bellman_ford_kernel");

    // TODO: should this memcpy go inside the loop?
    hipMemcpy(distance, deviceDistance, graph->nnode * sizeof(int), hipMemcpyDeviceToHost);
    // cudaCheckErrors("bellman_ford hipMemcpyDeviceToHost");

    hipFree(deviceSrcNodes);
    hipFree(deviceDstNodes);
    hipFree(deviceDistance);
    free(srcNodes);
    free(dstNodes);
    /**************************************************************/

    // Reweight edge weights
    for (int u = 0; u < graph->nnode; u++)
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            graph->new_weight[eid] = graph->weight[eid] + distance[u] - distance[v];
            if (graph->new_weight[eid] < 0) {
                printf("Graph contains negative weight cycle\n");
                exit(0);
            }
        }
}

__host__ void dijkstra_host(Graph *graph) {
    int* deviceNewWeights;
    int* deviceDistance;
    int* devicePredecessor;
    char* deviceVisited;
    int* deviceTmpDistance;

    int nnode = graph->nnode;
    int nedge = graph->nedge;

    hipMalloc(&deviceNewWeights, nedge * sizeof(int));
    hipMalloc(&deviceDistance, nnode * nnode * sizeof(int));
    hipMalloc(&deviceTmpDistance, nnode * nnode * sizeof(int));
    hipMalloc(&devicePredecessor, nnode * nnode * sizeof(int));
    hipMalloc(&deviceVisited, nnode * nnode * sizeof(char));
    // cudaCheckErrors("dijkstra hipMalloc");

    hipMemcpy(deviceNewWeights, graph->new_weight, sizeof(int) * nedge, hipMemcpyHostToDevice);
    // cudaCheckErrors("dijkstra hipMemcpyHostToDevice");

    int threadsPerBlock = 512;
    int blocks = (nnode + threadsPerBlock - 1) / threadsPerBlock;
    dijkstra_kernel<<<blocks, threadsPerBlock>>>(deviceNewWeights, deviceDistance, deviceTmpDistance, devicePredecessor, deviceVisited);
    hipDeviceSynchronize();
    // cudaCheckErrors("dijkstra_kernel");

    hipMemcpy(graph->distance, deviceDistance, nnode * nnode * sizeof(int), hipMemcpyDeviceToHost);
    // cudaCheckErrors("dijkstra hipMemcpyDeviceToHost");

    hipFree(deviceNewWeights);
    hipFree(deviceDistance);
    hipFree(deviceTmpDistance);
    hipFree(devicePredecessor);
    hipFree(deviceVisited);
}

__host__ void johnson_host(Graph *graph) {
    START_ACTIVITY(ACTIVITY_OVERHEAD);
    int* deviceNodes;
    int* deviceEdges;
    int* deviceWeights;

    int nnode = graph->nnode;
    int nedge = graph->nedge;

    hipMalloc(&deviceNodes, (nnode + 1) * sizeof(int));
    hipMalloc(&deviceEdges, nedge * sizeof(int));
    hipMalloc(&deviceWeights, nedge * sizeof(int));
    // cudaCheckErrors("johnson hipMalloc");

    hipMemcpy(deviceNodes, graph->node, sizeof(int) * (nnode + 1), hipMemcpyHostToDevice);
    hipMemcpy(deviceEdges, graph->edge, sizeof(int) * nedge, hipMemcpyHostToDevice);
    hipMemcpy(deviceWeights, graph->weight, sizeof(int) * nedge, hipMemcpyHostToDevice);
    // cudaCheckErrors("johnson hipMemcpyHostToDevice");

    GlobalConstants graphParams;
    graphParams.nnode = nnode;
    graphParams.nedge = nedge;
    graphParams.node = deviceNodes;
    graphParams.edge = deviceEdges;
    graphParams.weight = deviceWeights;

    hipMemcpyToSymbol(HIP_SYMBOL(constGraphParams), &graphParams, sizeof(GlobalConstants));
    // cudaCheckErrors("johnson hipMemcpyToSymbol");

    FINISH_ACTIVITY(ACTIVITY_OVERHEAD);

    // bellman_ford
    START_ACTIVITY(BELLMAN_FORD);
    bellman_ford_host(graph);
    FINISH_ACTIVITY(BELLMAN_FORD);

    // dijkstra
    START_ACTIVITY(DIJKSTRA);
    dijkstra_host(graph);
    FINISH_ACTIVITY(DIJKSTRA);

    hipFree(deviceNodes);
    hipFree(deviceEdges);
    hipFree(deviceWeights);
}

static void Usage(char *name) {
    char use_string[] = "-g GFILE [-v]";
    printf("Usage: %s %s\n", name, use_string);
    printf("   -h        Print this message\n");
    printf("   -g GFILE  Graph file\n");
    printf("   -v        Operate in verbose mode\n");
    exit(0);
}

int main(int argc, char *argv[]) {
    // Initialize cuda kernel mode driver
    hipFree(0);

    int c;
    FILE *graph_file = NULL;
    Graph *graph;
    display = 0;
    bool instrument = false;
    bool showMem = false;
    bool doPrint = false;

    // parse command line arguments
    while ((c = getopt(argc, argv, "hg:vIMP")) != -1) {
        switch(c) {
            case 'g':
                graph_file = fopen(optarg, "r");
                if (graph_file == NULL)
                    printf("Couldn't open graph file %s\n", optarg);
                break;
            case 'v':
                display = 1;
                break;
            case 'h':
                Usage(argv[0]);
                break;
            case 'I':
                instrument = true;
                break;
            case 'M':
                showMem = true;
                break;
            case 'P':
                doPrint = true;
                break;
            default:
                printf("Unknown option '%c'\n", c);
                Usage(argv[0]);
        }
    }

    if (showMem) {
        int deviceCount;
        hipDeviceProp_t deviceProp;
        hipGetDeviceCount(&deviceCount);
        for (int d = 0; d < deviceCount; ++d) {
            hipGetDeviceProperties(&deviceProp, d);
            std::cout
                << "***Device " << d << "***\n"
                << "Name: " << deviceProp.name << "\n"
        	      << "Total Global Memory (kB): " << deviceProp.totalGlobalMem / 1024 << "\n"
                << "Shared Memory per Block: " << deviceProp.sharedMemPerBlock << "\n"
                << "Total Constant Memory (B): " << deviceProp.totalConstMem << "\n"
                << "L2 Cache (B): " << deviceProp.l2CacheSize << "\n";
        }
        return 0;
    }

    track_activity(instrument);

    if (graph_file == NULL) {
	      printf("Need graph file\n");
        Usage(argv[0]);
        return 0;
    }

    START_ACTIVITY(LOAD_GRAPH);
    graph = LoadGraph(graph_file);
    FINISH_ACTIVITY(LOAD_GRAPH);

    johnson_host(graph);

    if (doPrint) {
        // output
        START_ACTIVITY(PRINT_GRAPH);
        for (int i = 0; i < graph->nnode; ++i) {
          for (int j = 0; j < graph->nnode; ++j) {
            if (graph->distance[i * graph->nnode + j] == IntMax)
                std::cout << std::setw(5) << "inf";
            else
                std::cout << std::setw(5) << graph->distance[i * graph->nnode + j];
          }
          std::cout << std::endl;
        }
        FINISH_ACTIVITY(PRINT_GRAPH);
    }

    SHOW_ACTIVITY(stderr, instrument);

    freeGraph(graph);
}
