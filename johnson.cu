#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

#define MaxLineLength 1024
#define IntMax __INT32_MAX__
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

char display;

typedef struct {
    int nnode;
    int nedge;

    int *node;
    int *edge;
    int *weight;
    int *new_weight;

    int **distance;
    int **predecessor;
} Graph;

Graph *LoadGraph(FILE *graph_file) {
    Graph *graph = (Graph *)malloc(sizeof(Graph));
    char linebuf[MaxLineLength];
    int src_id, dst_id, weight;
    int prev_src_id = 0;
    int lineno = 0;

    //  Load number of nodes and edges
    fgets(linebuf, MaxLineLength, graph_file);
    if (sscanf(linebuf, "%d", &graph->nnode) < 1) {
        printf("ERROR. Malformed graph file header (line 1)\n");
        return NULL;
    }

    fgets(linebuf, MaxLineLength, graph_file);
    if (sscanf(linebuf, "%d", &graph->nedge) < 1) {
        printf("ERROR. Malformed graph file header (line 1)\n");
        return NULL;
    }

    //  Initialize graph
    graph->node = (int *)calloc(graph->nnode+1, sizeof(int));
    graph->node[graph->nnode] = graph->nedge;
    graph->edge = (int *)malloc(graph->nedge * sizeof(int));
    graph->weight = (int *)malloc(graph->nedge * sizeof(int));
    graph->new_weight = (int *)malloc(graph->nedge * sizeof(int));
    graph->distance = (int **)malloc(graph->nnode * sizeof(int *));
    graph->predecessor = (int **)malloc(graph->nnode * sizeof(int *));
    for (int nid = 0; nid < graph->nnode; nid++) {
        graph->distance[nid] = (int *)malloc(graph->nnode * sizeof(int));
        graph->predecessor[nid] = (int *)malloc(graph->nnode * sizeof(int));
    }

    //  Load edges
    while (fgets(linebuf, MaxLineLength, graph_file) != NULL) {
        if (sscanf(linebuf, "%d %d %d", &src_id, &dst_id, &weight) < 3) {
            printf("ERROR. Malformed graph file header (line 1)\n");
            return NULL;
        }

        if (prev_src_id != src_id) {
            for (int i = prev_src_id+1; i <= src_id; i++)
                graph->node[i] = lineno;
            prev_src_id = src_id;
        }

        graph->edge[lineno] = dst_id;
        graph->weight[lineno] = weight;

        lineno++;
    }

    // Pad all later nodes with 0 out degree
    for (int nid = src_id+1; nid < graph->nnode; nid++) graph->node[nid] = graph->nedge;

    return graph;
}

///////////////////////////////////////////////////////////////////////////////
// Start of kernels
///////////////////////////////////////////////////////////////////////////////

__device__ __inline__ void BellmanFord(Graph *graph, int nid) {
    extern __shared__ int distance[];
    if (nid < graph->nnode)
        distance[nid] = 0;
    
    __syncthreads();

    for (int u = 0; u < graph->nnode; u++)
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            int weight = graph->weight[eid];
            if (distance[v] > distance[u] + weight)
                distance[v] = distance[u] + weight;
        }
    
    __syncthreads();

    if (nid < graph->nnode) {
        int u = nid;
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            graph->new_weight[eid] = graph->weight[eid] + distance[u] - distance[v];
            if (graph->new_weight[eid] < 0) {
                printf("Graph contains negative weight cycle\n");
                return;
            }
        }
    }
}

// Recursively calculate original weights
__device__ __inline__ void CalculateOriginalDistance(int src_nid, int nid, int *distance, int *predecessor, Graph *graph) {
    int current_nid = nid;
    int prev_nid = predecessor[current_nid];

    if (distance[nid] != -1)   // Distance is already alculated
        return;
    else if (nid == src_nid)    // This is the source node
        distance[nid] = 0;
    else if (predecessor[nid] == -1)    // No valid path to this node exists
        distance[nid] = IntMax;
    else {
        if (distance[prev_nid] == -1)
            CalculateOriginalDistance(src_nid, prev_nid, distance, predecessor, graph);
        // Distance increment by original edge weight
        for (int eid = graph->node[prev_nid]; eid < graph->node[prev_nid+1]; eid++)
            if (graph->edge[eid] == current_nid)
                distance[nid] = graph->weight[eid] + distance[prev_nid];
    }
}

// Functionality is explaned by function name
__device__ __inline__ int FindIndexOfUnvisitedNodeWithMinDistance(int nnode, int *distance, char *visited) {
    int min_nid = -1;
    int min_distance = IntMax;

    for (int nid = 0; nid < nnode; nid++)
        if (!visited[nid] && distance[nid] <= min_distance) {
            min_nid = nid;
            min_distance = distance[nid];
        }

    return min_nid;
}

__device__ __inline__ void Dijkstra(Graph *graph, int src_nid) {
    int *distance = graph->distance[src_nid];
    int *predecessor = graph->predecessor[src_nid];
    int *tmp_distance = (int *)malloc(graph->nnode * sizeof(int));
    char *visited = (char *)malloc(graph->nnode * sizeof(char));

    for (int nid = 0; nid < graph->nnode; nid++) {
        distance[nid] = -1;
        predecessor[nid] = -1;
        tmp_distance[nid] = IntMax;
        visited[nid] = 0;
    }
    tmp_distance[src_nid] = 0;
    predecessor[src_nid] = src_nid;

    for (int iter = 0; iter < graph->nnode; iter++) {
        int min_nid = FindIndexOfUnvisitedNodeWithMinDistance(graph->nnode, tmp_distance, visited);
        // No reachable unvisted nodes left
        if (tmp_distance[min_nid] == IntMax) break;

        visited[min_nid] = 1;
        for (int eid = graph->node[min_nid]; eid < graph->node[min_nid+1]; eid++) {
            int neighbor_nid = graph->edge[eid];
            if (tmp_distance[neighbor_nid] > graph->new_weight[eid] + tmp_distance[min_nid]) {
                tmp_distance[neighbor_nid] = graph->new_weight[eid] + tmp_distance[min_nid];
                predecessor[neighbor_nid] = min_nid;
            }
        }
    }

    for (int nid = 0; nid < graph->nnode; nid++)
        CalculateOriginalDistance(src_nid, nid, distance, predecessor, graph);

    free(tmp_distance);
    free(visited);
}

__global__ void KernelJohnson(Graph *device_graph) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index == 0) printf("%d\n", device_graph->node[3]);

    // BellmanFord(device_graph, index);
    // Dijkstra(device_graph, index);
}

///////////////////////////////////////////////////////////////////////////////
/// End of kernels
///////////////////////////////////////////////////////////////////////////////

static void Usage(char *name) {
    char use_string[] = "-g GFILE [-v]";
    printf("Usage: %s %s\n", name, use_string);
    printf("   -h        Print this message\n");
    printf("   -g GFILE  Graph file\n");
    printf("   -v        Operate in verbose mode\n");
    exit(0);
}

int main(int argc, char *argv[]) {
    int c;
    FILE *graph_file = NULL;
    Graph *graph;
    display = 0;
    bool instrument = false;

    // parse command line arguments
    while ((c = getopt(argc, argv, "hg:vI")) != -1) {
        switch(c) {
            case 'g':
                graph_file = fopen(optarg, "r");
                if (graph_file == NULL)
                    printf("Couldn't open graph file %s\n", optarg);
                break;
            case 'v':
                display = 1;
                break;
            case 'h':
                Usage(argv[0]);
                break;
            case 'I':
                instrument = true;
                break;
            default:
                printf("Unknown option '%c'\n", c);
                Usage(argv[0]);
        }
    }

    if (graph_file == NULL) {
	    printf("Need graph file\n");
        Usage(argv[0]);
        return 0;
    }

    graph = LoadGraph(graph_file);

    Graph *device_graph;
    hipMalloc(&device_graph, sizeof(graph));
    // cudaMalloc(&device_graph->node, graph->nnode * sizeof(int));
    // cudaMalloc(&device_graph->edge, graph->nedge * sizeof(int));
    // cudaMalloc(&device_graph->weight, graph->nedge * sizeof(int));
    // cudaMalloc(&device_graph->new_weight, graph->nedge * sizeof(int));
    // cudaMalloc(&device_graph->distance, graph->nnode * sizeof(int *));
    // cudaMalloc(&device_graph->predecessor, graph->nnode * sizeof(int *));
    // for (int nid = 0; nid < graph->nnode; nid++) {
    //     cudaMalloc(&device_graph->distance[nid], graph->nnode * sizeof(int));
    //     cudaMalloc(&device_graph->predecessor[nid], graph->nnode * sizeof(int));
    // }
    
    hipMemcpy(device_graph, graph, sizeof(Graph), hipMemcpyHostToDevice);

    const int ThreadsPerBlock = 512;
    const int Blocks = (graph->nnode + ThreadsPerBlock - 1) / ThreadsPerBlock;

    KernelJohnson<<<Blocks, ThreadsPerBlock, graph->nnode>>>(device_graph);
    cudaCheckError(hipDeviceSynchronize());
    hipMemcpy(graph, device_graph, sizeof(graph), hipMemcpyDeviceToHost);
}