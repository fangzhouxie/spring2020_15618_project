#include <iomanip>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

#define MaxLineLength 1024
#define IntMax __INT32_MAX__
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

char display;

struct GlobalConstants {
    int nnode;
    int nedge;

    int *node;
    int *edge;
    int *weight;
};

typedef struct {
    int nnode;
    int nedge;

    int *node;
    int *edge;
    int *weight;
    int *new_weight;

    int *distance;
    int *predecessor;
} Graph;

__constant__ GlobalConstants constGraphParams;

Graph *LoadGraph(FILE *graph_file) {
    Graph *graph = (Graph *)malloc(sizeof(Graph));
    char linebuf[MaxLineLength];
    int src_id, dst_id, weight;
    int prev_src_id = 0;
    int lineno = 0;

    //  Load number of nodes and edges
    fgets(linebuf, MaxLineLength, graph_file);
    if (sscanf(linebuf, "%d", &graph->nnode) < 1) {
        printf("ERROR. Malformed graph file header (line 1)\n");
        return NULL;
    }

    fgets(linebuf, MaxLineLength, graph_file);
    if (sscanf(linebuf, "%d", &graph->nedge) < 1) {
        printf("ERROR. Malformed graph file header (line 1)\n");
        return NULL;
    }

    //  Initialize graph
    graph->node = (int *)calloc(graph->nnode+1, sizeof(int));
    graph->node[graph->nnode] = graph->nedge;
    graph->edge = (int *)malloc(graph->nedge * sizeof(int));
    graph->weight = (int *)malloc(graph->nedge * sizeof(int));
    graph->new_weight = (int *)malloc(graph->nedge * sizeof(int));
    graph->distance = (int *)malloc(graph->nnode * graph->nnode * sizeof(int));
    graph->predecessor = (int *)malloc(graph->nnode * graph->nnode * sizeof(int));

    //  Load edges
    while (fgets(linebuf, MaxLineLength, graph_file) != NULL) {
        if (sscanf(linebuf, "%d %d %d", &src_id, &dst_id, &weight) < 3) {
            printf("ERROR. Malformed graph file header (line 1)\n");
            return NULL;
        }

        if (prev_src_id != src_id) {
            for (int i = prev_src_id+1; i <= src_id; i++)
                graph->node[i] = lineno;
            prev_src_id = src_id;
        }

        graph->edge[lineno] = dst_id;
        graph->weight[lineno] = weight;

        lineno++;
    }

    // Pad all later nodes with 0 out degree
    for (int nid = src_id+1; nid < graph->nnode; nid++) graph->node[nid] = graph->nedge;

    return graph;
}

///////////////////////////////////////////////////////////////////////////////
// Start of kernels
///////////////////////////////////////////////////////////////////////////////

__device__ __inline__ void BellmanFord(Graph *graph, int nid) {
    extern __shared__ int distance[];
    if (nid < graph->nnode)
        distance[nid] = 0;

    __syncthreads();

    for (int u = 0; u < graph->nnode; u++)
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            int weight = graph->weight[eid];
            if (distance[v] > distance[u] + weight)
                distance[v] = distance[u] + weight;
        }

    __syncthreads();

    if (nid < graph->nnode) {
        int u = nid;
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            graph->new_weight[eid] = graph->weight[eid] + distance[u] - distance[v];
            if (graph->new_weight[eid] < 0) {
                printf("Graph contains negative weight cycle\n");
                return;
            }
        }
    }
}

// Recursively calculate original weights
__device__ __inline__ void CalculateOriginalDistance(int src_nid, int nid, int *distance, int *predecessor) {
    int current_nid = nid;
    int prev_nid = predecessor[current_nid];

    int *node = constGraphParams.node;
    int *edge = constGraphParams.edge;
    int *weight = constGraphParams.weight;

    if (distance[nid] != -1)   // Distance is already alculated
        return;
    else if (nid == src_nid)    // This is the source node
        distance[nid] = 0;
    else if (predecessor[nid] == -1)    // No valid path to this node exists
        distance[nid] = IntMax;
    else {
        if (distance[prev_nid] == -1)
            CalculateOriginalDistance(src_nid, prev_nid, distance, predecessor);
        // Distance increment by original edge weight
        for (int eid = node[prev_nid]; eid < node[prev_nid+1]; eid++)
            if (edge[eid] == current_nid)
                distance[nid] = weight[eid] + distance[prev_nid];
    }
}

// Functionality is explaned by function name
__device__ __inline__ int FindIndexOfUnvisitedNodeWithMinDistance(int nnode, int *distance, char *visited) {
    int min_nid = -1;
    int min_distance = IntMax;

    for (int nid = 0; nid < nnode; nid++)
        if (!visited[nid] && distance[nid] <= min_distance) {
            min_nid = nid;
            min_distance = distance[nid];
        }

    return min_nid;
}

__global__ void dijkstra_kernel(int* new_weight, int* distance) {
    int src_nid = blockIdx.x * blockDim.x + threadIdx.x;
    int nnode = constGraphParams.nnode;
    if (src_nid >= nnode) return;

    int *node = constGraphParams.node;
    int *edge = constGraphParams.edge;

    int *distance_local = &distance[src_nid * nnode];
    int *predecessor_local = (int *)malloc(nnode * sizeof(int));
    int *tmp_distance = (int *)malloc(nnode * sizeof(int));
    char *visited_local = (char *)malloc(nnode * sizeof(char));

    for (int nid = 0; nid < nnode; nid++) {
        distance_local[nid] = -1;
        predecessor_local[nid] = -1;
        tmp_distance[nid] = IntMax;
        visited_local[nid] = 0;
    }
    tmp_distance[src_nid] = 0;
    predecessor_local[src_nid] = src_nid;

    for (int iter = 0; iter < nnode; iter++) {
        int min_nid = FindIndexOfUnvisitedNodeWithMinDistance(nnode, tmp_distance, visited_local);
        // No reachable unvisted nodes left
        if (tmp_distance[min_nid] == IntMax) break;

        visited_local[min_nid] = 1;
        for (int eid = node[min_nid]; eid < node[min_nid+1]; eid++) {
            int neighbor_nid = edge[eid];
            if (tmp_distance[neighbor_nid] > new_weight[eid] + tmp_distance[min_nid]) {
                tmp_distance[neighbor_nid] = new_weight[eid] + tmp_distance[min_nid];
                predecessor_local[neighbor_nid] = min_nid;
            }
        }
    }

    for (int nid = 0; nid < nnode; nid++)
        CalculateOriginalDistance(src_nid, nid, distance_local, predecessor_local);

    free(tmp_distance);
    free(visited_local);
    free(predecessor_local);
}

///////////////////////////////////////////////////////////////////////////////
/// End of kernels
///////////////////////////////////////////////////////////////////////////////

//TODO: bellman ford is sequential for now
__host__ void bellman_ford_host(Graph *graph) {
    int distance[graph->nnode];

    // Initialize distances from new source node to all nodes
    for (int nid = 0; nid < graph->nnode; nid++)
        distance[nid] = 0;

    // Iterate through the graph V - 1 times
    for (int iter = 0; iter < graph->nnode; iter++)
        for (int u = 0; u < graph->nnode; u++)
            for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
                int v = graph->edge[eid];
                int weight = graph->weight[eid];
                if (distance[v] > distance[u] + weight)
                    distance[v] = distance[u] + weight;
            }

    // Reweight edge weights
    for (int u = 0; u < graph->nnode; u++)
        for (int eid = graph->node[u]; eid < graph->node[u+1]; eid++) {
            int v = graph->edge[eid];
            graph->new_weight[eid] = graph->weight[eid] + distance[u] - distance[v];
            if (graph->new_weight[eid] < 0) {
                printf("Graph contains negative weight cycle\n");
                exit(0);
            }
        }
}

__host__ void dijkstra_host(Graph *graph) {
    int* deviceNewWeights;
    int* deviceDistance; // is it needed?
    //int* devicePredecessor; // is it needed?
    //char* deviceVisited;

    int nnode = graph->nnode;
    int nedge = graph->nedge;

    hipMalloc(&deviceNewWeights, nedge * sizeof(int));
    hipMalloc(&deviceDistance, nnode * nnode * sizeof(int));

    hipMemcpy(deviceNewWeights, graph->new_weight, sizeof(int) * nedge, hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    int blocks = (nnode + threadsPerBlock - 1) / threadsPerBlock;
    dijkstra_kernel<<<blocks, threadsPerBlock>>>(deviceNewWeights, deviceDistance);

    hipMemcpy(graph->distance, deviceDistance, nnode * nnode * sizeof(int), hipMemcpyDeviceToHost);
}

__host__ void johnson_host(Graph *graph) {
    int* deviceNodes;
    int* deviceEdges;
    int* deviceWeights;

    int nnode = graph->nnode;
    int nedge = graph->nedge;

    hipMalloc(&deviceNodes, (nnode + 1) * sizeof(int));
    hipMalloc(&deviceEdges, nedge * sizeof(int));
    hipMalloc(&deviceWeights, nedge * sizeof(int));

    hipMemcpy(deviceNodes, graph->node, sizeof(int) * nnode, hipMemcpyHostToDevice);
    hipMemcpy(deviceEdges, graph->edge, sizeof(int) * nedge, hipMemcpyHostToDevice);
    hipMemcpy(deviceWeights, graph->weight, sizeof(int) * nedge, hipMemcpyHostToDevice);

    GlobalConstants graphParams;
    graphParams.nnode = nnode;
    graphParams.nedge = nedge;
    graphParams.node = deviceNodes;
    graphParams.edge = deviceEdges;
    graphParams.weight = deviceWeights;

    hipMemcpyToSymbol(HIP_SYMBOL(constGraphParams), &graphParams, sizeof(GlobalConstants));

    // bellman_ford
    bellman_ford_host(graph);

    // dijkstra
    dijkstra_host(graph);
}

static void Usage(char *name) {
    char use_string[] = "-g GFILE [-v]";
    printf("Usage: %s %s\n", name, use_string);
    printf("   -h        Print this message\n");
    printf("   -g GFILE  Graph file\n");
    printf("   -v        Operate in verbose mode\n");
    exit(0);
}

int main(int argc, char *argv[]) {
    int c;
    FILE *graph_file = NULL;
    Graph *graph;
    display = 0;
    bool instrument = false;

    // parse command line arguments
    while ((c = getopt(argc, argv, "hg:vI")) != -1) {
        switch(c) {
            case 'g':
                graph_file = fopen(optarg, "r");
                if (graph_file == NULL)
                    printf("Couldn't open graph file %s\n", optarg);
                break;
            case 'v':
                display = 1;
                break;
            case 'h':
                Usage(argv[0]);
                break;
            case 'I':
                instrument = true;
                break;
            default:
                printf("Unknown option '%c'\n", c);
                Usage(argv[0]);
        }
    }

    if (graph_file == NULL) {
	    printf("Need graph file\n");
        Usage(argv[0]);
        return 0;
    }

    graph = LoadGraph(graph_file);

    johnson_host(graph);

    // output
    for (int i = 0; i < graph->nnode; ++i) {
      for (int j = 0; j < graph->nnode; ++j) {
        if (graph->distance[i * graph->nnode + j] == IntMax)
            std::cout << std::setw(5) << "inf";
        else
            std::cout << std::setw(5) << graph->distance[i * graph->nnode + j];
      }
      std::cout << std::endl;
    }
}
